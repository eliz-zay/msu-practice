#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <map>
#include <string>
#include <vector>
#include <stdbool.h>
#include <tuple>
#include <unistd.h>
#include <sys/types.h>
#include <dirent.h>
#include <stdio.h>

#include "../external/lodepng/lodepng.cpp"

#define sqr(a) ((a)*(a))

using namespace std;

int maxGridDimX;
int maxGridDimY;

vector<string> listDir(const char* name) {
    DIR* dir;
    struct dirent* entry;
    vector<string> files;

    if (!(dir = opendir(name)))
        return files;

    while ((entry = readdir(dir)) != NULL) {
        if (strcmp(entry->d_name, ".") && strcmp(entry->d_name, "..")) {
            files.push_back(entry->d_name);
        }
    }
    closedir(dir);

    return files;
}

vector<vector<float> > kernels = {
    vector<float> { // gaussian 3x3
        1.0/16, 2.0/16, 1.0/16,
        2.0/16, 4.0/16, 2.0/16,
        1.0/16, 2.0/16, 1.0/16
    },
    vector<float> { // gaussian 5x5
        1.0/256, 4.0/256,   6.0/256,  4.0/256, 1.0/256,
        4.0/256, 16.0/256, 24.0/256, 16.0/256, 4.0/256,
        6.0/256, 24.0/256, 36.0/256, 24.0/256, 6.0/256,
        4.0/256, 16.0/256, 24.0/256, 16.0/256, 4.0/256,
        1.0/256, 4.0/256,   6.0/256,  4.0/256, 1.0/256
    },
    vector<float> { // edge detect 3x3
        -1, -1, -1,
        -1,  8, -1,
        -1, -1, -1
    }
};

void getError(hipError_t err) {
    if (err != hipSuccess) {
        printf("CUDA error - %s\n", hipGetErrorString(err));
    }
}

__global__ void apply_kernel_device(
    unsigned char* input_image,
    unsigned char* output_image,
    int width,
    int height,
    float* kernel,
    char kernel_dim
) {
    const unsigned int linearX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int linearY = blockIdx.y * blockDim.y + threadIdx.y;

    if (linearX >= width || linearY >= height) {
        return;
    }

    float r = 0, g = 0, b = 0;

    // Ядро 3х3, отсекаем рамку из 1 пикселя
    if (kernel_dim == 3 && linearX > 0 && linearX < width - 1 && linearY > 0 && linearY < height - 1) {
        for (int i = -1; i < 2; i++) {
            for (int j = -1; j < 2; j++) {
                r += input_image[3 * ((linearY + i) * width + (linearX + j))] * kernel[3 * (i + 1) + j + 1];
                g += input_image[3 * ((linearY + i) * width + (linearX + j)) + 1] * kernel[3 * (i + 1) + j + 1];
                b += input_image[3 * ((linearY + i) * width + (linearX + j)) + 2] * kernel[3 * (i + 1) + j + 1];
            }
        }

        output_image[3 * (linearY * width + linearX)] = ceil(r);
        output_image[3 * (linearY * width + linearX) + 1] = ceil(g);
        output_image[3 * (linearY * width + linearX) + 2] = ceil(b);

    // Ядро 5х5, отсекаем рамку из 2 пикселей
    } else if ((kernel_dim == 5 && linearX > 1 && linearX < width - 2 && linearY > 2 && linearY < height - 2)) {
        for (int i = -2; i < 3; i++) {
            for (int j = -2; j < 3; j++) {
                r += input_image[3 * ((linearY + i) * width + (linearX + j))] * kernel[3 * (i + 2) + j + 2];
                g += input_image[3 * ((linearY + i) * width + (linearX + j)) + 1] * kernel[3 * (i + 2) + j + 2];
                b += input_image[3 * ((linearY + i) * width + (linearX + j)) + 2] * kernel[3 * (i + 2) + j + 2];
            }
        }

        output_image[3 * (linearY * width + linearX)] = ceil(r);
        output_image[3 * (linearY * width + linearX) + 1] = ceil(g);
        output_image[3 * (linearY * width + linearX) + 2] = ceil(b);

    // То что попало в рамки (соотв. из 1 и из 2 пикселей)
    } else {
        output_image[3 * (linearY * width + linearX)] = input_image[3 * (linearY * width + linearX)];
        output_image[3 * (linearY * width + linearX) + 1] = input_image[3 * (linearY * width + linearX) + 1];
        output_image[3 * (linearY * width + linearX) + 2] = input_image[3 * (linearY * width + linearX) + 2];
    }
}

void apply_kernel(unsigned char* input_image, unsigned char* output_image, int width, int height, int kernelId, float& calcTime, float& totalTime) {
    unsigned char* dev_input;
    unsigned char* dev_output;
    float* dev_kernel;

    int size = kernels[kernelId].size();
    float* kernel = kernels[kernelId].data();

    float ms_outer = 0;
    float ms_inner = 0;
    hipEvent_t start_outer;
    hipEvent_t stop_outer;
    hipEvent_t start_inner;
    hipEvent_t stop_inner;
    hipEventCreate(&start_outer);
    hipEventCreate(&stop_outer);
    hipEventCreate(&start_inner);
    hipEventCreate(&stop_inner);

    hipEventRecord(start_outer);
    hipEventSynchronize(start_outer);

    getError(hipMalloc((void **)&dev_input, 3 * width * height * sizeof(unsigned char)));
    getError(hipMemcpy(dev_input, input_image, 3 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice));

    getError(hipMalloc((void **)&dev_kernel, size * sizeof(float)));
    getError(hipMemcpy(dev_kernel, kernel, size * sizeof(float), hipMemcpyHostToDevice));

    getError(hipMalloc((void **)&dev_output, 3 * width * height * sizeof(unsigned char)));

    int blockDim = 32;
    int gridDimX = ceil(1.0 * width / blockDim);
    int gridDimY = ceil(1.0 * height / blockDim);

    if (gridDimX > maxGridDimX || gridDimY > maxGridDimY) {
        throw runtime_error("Error! Image too big");
    }

    printf("Sizes: block size %d, grid x-dim %d, grid y-dim %d\n", blockDim, gridDimX, gridDimY);

    dim3 blockDims(blockDim, blockDim, 1);
    dim3 gridDims(gridDimX, gridDimY, 1);

    hipEventRecord(start_inner);
    hipEventSynchronize(start_inner);

    apply_kernel_device<<<gridDims, blockDims>>>(dev_input, dev_output, width, height, dev_kernel, (int)(sqrt(size)));

    hipEventRecord(stop_inner);
    hipEventSynchronize(stop_inner);
    hipEventElapsedTime(&ms_inner, start_inner, stop_inner);

    getError(hipMemcpy(output_image, dev_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

    getError(hipFree(dev_input));
    getError(hipFree(dev_output));
    getError(hipFree(dev_kernel));

    hipEventRecord(stop_outer);
    hipEventSynchronize(stop_outer);
    hipEventElapsedTime(&ms_outer, start_outer, stop_outer);

    printf("GPU calculation time: %g ms\n", ms_inner);
    printf("GPU total time: %g ms\n", ms_outer);

    calcTime = ms_inner;
    totalTime = ms_outer;
}

void loadCudaSettings() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    maxGridDimX = prop.maxGridSize[0];
    maxGridDimY = prop.maxGridSize[1];
}

void processImage(char* inFile, char* outFile, int kernel, float& calcTime, float& totalTime) {
    vector<unsigned char> rgbaIn;
    unsigned int width, height;

    unsigned error = lodepng::decode(rgbaIn, width, height, inFile);
    if (error) {
        cout << "decoder error " << error << ": " << lodepng_error_text(error) << endl;
    }

    unsigned char *rgbIn = new unsigned char[(rgbaIn.size() * 3) / 4];
    unsigned char *rgbOut = new unsigned char[(rgbaIn.size() * 3) / 4];
    int inp_iterator = 0;
    for (int i = 0; i < rgbaIn.size(); ++i) {
        if ((i + 1) % 4 != 0) { // 3,7... - под альфа пропускаем
            rgbIn[inp_iterator] = rgbaIn.at(i);
            rgbOut[inp_iterator] = 255;
            inp_iterator++;
        }
    }

    printf("Image size - %dx%d\n", width, height);

    apply_kernel(rgbIn, rgbOut, width, height, kernel, calcTime, totalTime);

    int out_iterator = 0;
    vector<unsigned char> rgbaOut(rgbaIn.size());
    for (int i = 0; i < width * height * 3; ++i) {
        rgbaOut[out_iterator] = rgbOut[i];
        out_iterator++;
        if ((i + 1) % 3 == 0) { // в конец каждой тройки rgb пишем альфа
            rgbaOut[out_iterator] = 255;
            out_iterator++;
        }
    }

    error = lodepng::encode(outFile, rgbaOut, width, height);

    if (error) {
        printf("Encoder error: %s\n", lodepng_error_text(error));
    }

    delete[] rgbIn;
    delete[] rgbOut;
}

void parseArgs(int argc, char** argv, int* kernel, char** imgType) {
    if (argc != 3) {
        cout << "2 arguments required" << endl;
        exit(0);
    }

    *kernel = atoi(argv[1]);
    *imgType = argv[2];

    if (*kernel < 0 || *kernel > 2) {
        cout << "Kernel idx must be in range [0,2]" << endl;
        exit(0);
    }
}

int main(int argc, char** argv) {
    vector<pair<char*, char*> > images;
    char* imgType;
    int kernel;

    parseArgs(argc, argv, &kernel, &imgType);

    if (!strcmp(imgType,"big")) {
        images.push_back({ "in/big.png", "out/big.png" });

    } else if (!strcmp(imgType,"small")) {
        vector<string> files = listDir("in/small/");
        for (string name: files) {
            string strIn = "in/small/" + name;
            string strOut = "out/" + name;
            char *in = new char[strIn.length() + 1];
            char *out = new char[strOut.length() + 1];
            strcpy(in, strIn.c_str());
            strcpy(out, strOut.c_str());
            images.push_back({ in, out });
        }
    
    } else {
        cout << "Invalid arguments" << endl;
        return 0;
    }

    loadCudaSettings();

    float calcTimeSum = 0, totalTimeSum = 0;

    for (int i = 0; i < images.size(); i++) {
        printf("Started processing image %s\n", images[i].first);
        float calcTime = 0, totalTime = 0;
        processImage(images[i].first, images[i].second, kernel, calcTime, totalTime);
        calcTimeSum += calcTime;
        totalTimeSum += totalTime;
        printf("Finised. Output was written to %s\n", images[i].second);
    }

    printf("Sum of calculation times - %g\n", calcTimeSum);
    printf("Sum of total times - %g\n", totalTimeSum);
    
    return 0;
}